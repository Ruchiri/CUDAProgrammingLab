#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <tuple>
#include <iostream>
#include <string.h>

double time_host = 0;
double time_device = 0;

int sample_rounds = 10;

void meanFilter_host(unsigned char* image_matrix,unsigned char* filtered_image_data,int image_width, int image_height, int window_size)
{
    int half_window = (window_size-window_size % 2)/2;
	
    for(int i = 0; i < image_height; i += 1){
        for(int j = 0; j < image_width; j += 1){
            int k = 3*(i*image_height+j);
            int top_boundary;
			int bottom_boundary;
			int left_boundary;
			int right_boundary; 
            if(i-half_window >= 0){
				top_boundary = i-half_window;
			}else{
				top_boundary = 0;
			}
            if(i+half_window <= image_height-1){
				bottom_boundary = i+half_window;
			}else{
				bottom_boundary = image_height-1;
			}
            if(j-half_window >= 0){
				left_boundary = j-half_window;
			}else{
				left_boundary = 0;
			}
            if(j+half_window <= image_width-1){
				right_boundary = j+half_window;
			}else{
				right_boundary = image_width-1;
			}
            double byte1 = 0; 
            double byte2 = 0; 
            double byte3 = 0; 
            
            for(int x = top_boundary; x <= bottom_boundary; x++){
                for(int y = left_boundary; y <= right_boundary; y++){
                    int pos = 3*(x*image_height + y); 
                    byte1 += image_matrix[pos];
                    byte2 += image_matrix[pos+1];
                    byte3 += image_matrix[pos+2];
                }
            }
            int effective_window_size = (bottom_boundary-top_boundary+1)*(right_boundary-left_boundary+1);
            filtered_image_data[k] = byte1/effective_window_size;
            filtered_image_data[k+1] = byte2/effective_window_size;
            filtered_image_data[k+2] = byte3/effective_window_size;

            
        }
    }
   
}

__global__ void meanFilter_device(unsigned char* image_matrix, unsigned char* filtered_image_data, int image_width, int image_height, int window_size)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
	int half_window = (window_size-window_size % 2)/2;
	
    if (i < image_height && j < image_width){
        int k = 3*(i*image_height+j);
        int top_boundary;
		int bottom_boundary;
		int left_boundary;
		int right_boundary; 
        if(i-half_window >= 0){
			top_boundary = i-half_window;
		}else{
			top_boundary = 0;
		}
        if(i+half_window <= image_height-1){
			bottom_boundary = i+half_window;
		}else{
			bottom_boundary = image_height-1;
		}
        if(j-half_window >= 0){
			left_boundary = j-half_window;
		}else{
			left_boundary = 0;
		}
        if(j+half_window <= image_width-1){
			right_boundary = j+half_window;
		}else{
			right_boundary = image_width-1;
		}
        double byte1 = 0; 
        double byte2 = 0; 
        double byte3 = 0; 
       
        for(int x = top_boundary; x <= bottom_boundary; x++){
            for(int y = left_boundary; y <= right_boundary; y++){
                int pos = 3*(x*image_height + y); 
                byte1 += image_matrix[pos];
                byte2 += image_matrix[pos+1];
                byte3 += image_matrix[pos+2];
            }
        }
        int effective_window_size = (bottom_boundary-top_boundary+1)*(right_boundary-left_boundary+1);
        filtered_image_data[k] = byte1/effective_window_size;
        filtered_image_data[k+1] = byte2/effective_window_size;
        filtered_image_data[k+2] = byte3/effective_window_size;
    }
}


int main(int argc,char **argv)
{
   
    FILE* f = fopen(argv[1], "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); 

    int width, height;
    memcpy(&width, info + 18, sizeof(int));
    memcpy(&height, info + 22, sizeof(int));

    int window_size = strtol(argv[2],NULL,10);
    printf("     Window size: %d\n",window_size);
    printf("Image dimensions: (%d, %d)\n",width,height);
        
    int size = 3 * width * abs(height);
    unsigned char* data = new unsigned char[size]; 
    unsigned char* result_image_data_d;
    unsigned char* result_image_data_h = new unsigned char[size];
    unsigned char* result_image_data_h1 = new unsigned char[size];

    unsigned char* image_data_d;

    fread(data, sizeof(unsigned char), size, f); 
    fclose(f);
   
    int block_size = 32;
    int grid_size = width/block_size;
    dim3 dimBlock(block_size, block_size, 1);
    dim3 dimGrid(grid_size, grid_size, 1);

    
    for(int i = 0; i < sample_rounds; i += 1)
    {
        hipMalloc((void **)&image_data_d,size*sizeof(unsigned char));
        hipMalloc((void **)&result_image_data_d,size*sizeof(unsigned char));
        hipMemcpy(image_data_d,data,size*sizeof(unsigned char),hipMemcpyHostToDevice);
        
       
        clock_t start_d=clock();
        meanFilter_device <<< dimGrid, dimBlock >>> (image_data_d, result_image_data_d, width, height, window_size);
        hipDeviceSynchronize();

        hipError_t error = hipGetLastError();
        if(error!=hipSuccess)
        {
            fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
            exit(-1);
        }
        clock_t end_d = clock();

        clock_t start_h = clock();
        meanFilter_host(data, result_image_data_h1, width, height, window_size);
        clock_t end_h = clock();

        hipMemcpy(result_image_data_h,result_image_data_d,size*sizeof(unsigned char),hipMemcpyDeviceToHost);

        time_host += (double)(end_h-start_h)/CLOCKS_PER_SEC;
        time_device += (double)(end_d-start_d)/CLOCKS_PER_SEC;

        hipFree(image_data_d);
        hipFree(result_image_data_d);
    }

    printf("    GPU Time: %f\n",(time_device/sample_rounds));
    printf("    CPU Time: %f\n",(time_host/sample_rounds));
    printf("CPU/GPU time: %f\n",(time_host/time_device));

    
    return 0;
}

